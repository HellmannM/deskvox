
#include <hip/hip_runtime.h>
//
// This software contains source code provided by NVIDIA Corporation.
//

#ifdef HAVE_CONFIG_H
#include "vvconfig.h"
#endif

#if defined(HAVE_CUDA) && defined(NV_PROPRIETARY_CODE)

#include "vvglew.h"

#include "vvcuda.h"
#include "vvcudaimg.h"
#include "vvcudautils.h"
#include "vvdebugmsg.h"
#include "vvgltools.h"
#include "vvrayrend.h"

#include <cuda_gl_interop.h>
#include <ctime>
#include <iostream>
#include <limits>

using std::cerr;
using std::endl;

texture<uchar, 3, cudaReadModeNormalizedFloat> volTexture8;
texture<ushort, 3, cudaReadModeNormalizedFloat> volTexture16;
texture<float4, 1, cudaReadModeElementType> tfTexture;
texture<float, 3, cudaReadModeElementType> spaceSkippingTexture;
texture<float4, 1, cudaReadModeElementType> randTexture;

const int NUM_RAND_VECS = 8192;

int iDivUp(const int a, const int b)
{
  return (a % b != 0) ? (a / b + 1) : (a / b);
}

typedef struct
{
  float m[4][4];
} matrix4x4;

__constant__ matrix4x4 c_invViewMatrix;
__constant__ matrix4x4 c_MvPrMatrix;

struct Ray
{
  float3 o;
  float3 d;
};

template<int t_bpc>
__device__ float volume(const float x, const float y, const float z)
{
  if (t_bpc == 1)
  {
    return tex3D(volTexture8, x, y, z);
  }
  else if (t_bpc == 2)
  {
    return tex3D(volTexture16, x, y, z);
  }
  else
  {
    return -1.0f;
  }
}

template<int t_bpc>
__device__ float volume(const float3& pos)
{
  if (t_bpc == 1)
  {
    return tex3D(volTexture8, pos.x, pos.y, pos.z);
  }
  else if (t_bpc == 2)
  {
    return tex3D(volTexture16, pos.x, pos.y, pos.z);
  }
  else
  {
    return -1.0f;
  }
}

__device__ bool skipSpace(const float3& pos)
{
  //return (tex3D(spaceSkippingTexture, pos.x, pos.y, pos.z) == 0.0f);
  return false;
}

__device__ float3 calcTexCoord(const float3& pos, const float3& volPos, const float3& volSizeHalf)
{
  return make_float3((pos.x - volPos.x + volSizeHalf.x) / (volSizeHalf.x * 2.0f),
                     (pos.y - volPos.y + volSizeHalf.y) / (volSizeHalf.y * 2.0f),
                     (pos.z - volPos.z + volSizeHalf.z) / (volSizeHalf.z * 2.0f));
}

__device__ bool solveQuadraticEquation(const float A, const float B, const float C,
                                       float* tnear, float* tfar)
{
  const float discrim = B * B - 4.0f * A * C;
  if (discrim < 0.0f)
  {
    *tnear = -1.0f;
    *tfar = -1.0f;
  }
  const float rootDiscrim = __fsqrt_rn(discrim);
  float q;
  if (B < 0)
  {
    q = -0.5f * (B - rootDiscrim);
  }
  else
  {
    q = -0.5f * (B + rootDiscrim);
  }
  *tnear = q / A;
  *tfar = C / q;
  if (*tnear > *tfar)
  {
    float tmp = *tnear;
    *tnear = *tfar;
    *tfar = tmp;
    return true;
  }
  return false;
}

__device__ bool intersectBox(const Ray& ray, const float3& boxmin, const float3& boxmax,
                             float* tnear, float* tfar)
{
  // compute intersection of ray with all six bbox planes
  float3 invR = make_float3(1.0f, 1.0f, 1.0f) / ray.d;
  float t1 = (boxmin.x - ray.o.x) * invR.x;
  float t2 = (boxmax.x - ray.o.x) * invR.x;
  float tmin = fminf(t1, t2);
  float tmax = fmaxf(t1, t2);

  t1 = (boxmin.y - ray.o.y) * invR.y;
  t2 = (boxmax.y - ray.o.y) * invR.y;
  tmin = fmaxf(fminf(t1, t2), tmin);
  tmax = fminf(fmaxf(t1, t2), tmax);

  t1 = (boxmin.z - ray.o.z) * invR.z;
  t2 = (boxmax.z - ray.o.z) * invR.z;
  tmin = fmaxf(fminf(t1, t2), tmin);
  tmax = fminf(fmaxf(t1, t2), tmax);

  *tnear = tmin;
  *tfar = tmax;

  return ((tmax >= tmin) && (tmax >= 0.0f));
}

__device__ bool intersectSphere(const Ray& ray, const float3& center, const float radiusSqr,
                                float* tnear, float* tfar)
{
  Ray r = ray;
  r.o -= center;
  float A = r.d.x * r.d.x + r.d.y * r.d.y
          + r.d.z * r.d.z;
  float B = 2 * (r.d.x * r.o.x + r.d.y * r.o.y
               + r.d.z * r.o.z);
  float C = r.o.x * r.o.x + r.o.y * r.o.y
          + r.o.z * r.o.z - radiusSqr;
  return solveQuadraticEquation(A, B, C, tnear, tfar);
}

__device__ void intersectPlane(const Ray& ray, const float3& normal, const float& dist,
                               float* nddot, float* tnear)
{
  *nddot = dot(normal, ray.d);
  const float vOrigin = dist - dot(normal, ray.o);
  *tnear = vOrigin / *nddot;
}


__device__ float4 mulPost(const matrix4x4& M, const float4& v)
{
  float4 result;
  result.x = M.m[0][0] * v.x + M.m[0][1] * v.y + M.m[0][2] * v.z + M.m[0][3] * v.w;
  result.y = M.m[1][0] * v.x + M.m[1][1] * v.y + M.m[1][2] * v.z + M.m[1][3] * v.w;
  result.z = M.m[2][0] * v.x + M.m[2][1] * v.y + M.m[2][2] * v.z + M.m[2][3] * v.w;
  result.w = M.m[3][0] * v.x + M.m[3][1] * v.y + M.m[3][2] * v.z + M.m[3][3] * v.w;
  return result;
}

__device__ float4 mulPre(const matrix4x4& M, const float4& v)
{
  float4 result;
  result.x = M.m[0][0] * v.x + M.m[1][0] * v.y + M.m[2][0] * v.z + M.m[3][0] * v.w;
  result.y = M.m[0][1] * v.x + M.m[1][1] * v.y + M.m[2][1] * v.z + M.m[3][1] * v.w;
  result.z = M.m[0][2] * v.x + M.m[1][2] * v.y + M.m[2][2] * v.z + M.m[3][2] * v.w;
  result.w = M.m[0][3] * v.x + M.m[1][3] * v.y + M.m[2][3] * v.z + M.m[3][3] * v.w;
  return result;
}

__device__ float3 perspectiveDivide(const float4& v)
{
  const float wInv = 1.0f / v.w;
  return make_float3(v.x * wInv, v.y * wInv, v.z * wInv);
}

__device__ uchar4 rgbaFloatToInt(float4 rgba)
{
  clamp(rgba.x);
  clamp(rgba.y);
  clamp(rgba.z);
  clamp(rgba.w);
  return make_uchar4(rgba.x * 255, rgba.y * 255,rgba.z * 255, rgba.w * 255);
}

template<int t_bpc>
__device__ float3 gradient(const float3& pos)
{
  const float DELTA = 0.01f;

  float3 sample1;
  float3 sample2;

  sample1.x = volume<t_bpc>(pos - make_float3(DELTA, 0.0f, 0.0f));
  sample2.x = volume<t_bpc>(pos + make_float3(DELTA, 0.0f, 0.0f));
  sample1.y = volume<t_bpc>(pos - make_float3(0.0f, DELTA, 0.0f));
  sample2.y = volume<t_bpc>(pos + make_float3(0.0f, DELTA, 0.0f));
  sample1.z = volume<t_bpc>(pos - make_float3(0.0f, 0.0f, DELTA));
  sample2.z = volume<t_bpc>(pos + make_float3(0.0f, 0.0f, DELTA));

  return sample2 - sample1;
}

template<int t_bpc>
__device__ float4 blinnPhong(const float4& classification, const float3& pos,
                             const float3& L, const float3& H,
                             const float3& Ka, const float3& Kd, const float3& Ks,
                             const float shininess,
                             const float3* normal = NULL)
{
  float3 N = normalize(gradient<t_bpc>(pos));

  if (normal != NULL)
  {
    // Interpolate gradient with normal from clip object (based on opacity).
    N = (*normal * classification.w) + (N * (1.0f - classification.w));
    N = normalize(N);
  }

  const float diffuse = fabsf(dot(L, N));
  const float specular = powf(dot(H, N), shininess);

  const float3 c = make_float3(classification);
  float3 tmp = Ka * c + Kd * diffuse * c;
  if (specular > 0.0f)
  {
    tmp += Ks * specular * c;
  }
  return make_float4(tmp.x, tmp.y, tmp.z, classification.w);
}



template<
         bool t_earlyRayTermination,
         bool t_spaceSkipping,
         bool t_frontToBack,
         int t_bpc,
         int t_mipMode,
         bool t_lighting,
         bool t_opacityCorrection,
         bool t_jittering,
         bool t_clipPlane,
         bool t_clipSphere,
         bool t_useSphereAsProbe
        >
__global__ void render(uchar4* d_output, const uint width, const uint height,
                       const float4 backgroundColor,
                       const uint texwidth, const float dist,
                       const float3 volPos, const float3 volSizeHalf,
                       const float3 probePos, const float3 probeSizeHalf,
                       const float3 L, const float3 H,
                       const float3 sphereCenter, const float sphereRadius,
                       const float3 planeNormal, const float planeDist,
                       void* d_depth, vvImage2_5d::DepthPrecision dp)
{
  const bool t_isaDepth = true;
  const int maxSteps = INT_MAX;
  const float opacityThreshold = 0.95f;

  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((x >= width) || (y >= height))
  {
    return;
  }

  const float u = (x / static_cast<float>(width)) * 2.0f - 1.0f;
  const float v = (y / static_cast<float>(height)) * 2.0f - 1.0f;

  /*
   * Rays like if the view were orthographic with origins over each pixel.
   * These are multiplied with the inverse modelview projection matrix.
   * First of all, the rays will be transformed to fit to the frustum.
   * Then the rays will be oriented so that they can hit the volume.
   */
  const float4 o = mulPost(c_invViewMatrix, make_float4(u, v, -1.0f, 1.0f));
  const float4 d = mulPost(c_invViewMatrix, make_float4(u, v, 1.0f, 1.0f));

  Ray ray;
  ray.o = perspectiveDivide(o);
  ray.d = perspectiveDivide(d);
  ray.d = ray.d - ray.o;
  ray.d = normalize(ray.d);

  float tnear;
  float tfar;
  const bool hit = intersectBox(ray, probePos - probeSizeHalf, probePos + probeSizeHalf, &tnear, &tfar);
  if (!hit)
  {
    d_output[y * texwidth + x] = make_uchar4(0);
    if(t_isaDepth)
    {
      switch(dp)
      {
      case vvImage2_5d::VV_UCHAR:
        ((unsigned char*)(d_depth))[y * texwidth + x] = 0;
        break;
      case vvImage2_5d::VV_USHORT:
        ((unsigned short*)(d_depth))[y * texwidth + x] = 0;
        break;
      case vvImage2_5d::VV_UINT:
        ((unsigned int*)(d_depth))[y * texwidth + x] = 0;
        break;
      }
    }
    return;
  }

  if (fmodf(tnear, dist) != 0.0f)
  {
    int tmp = (tnear / dist);
    tnear = dist * tmp;
  }

  if (tnear < 0.0f)
  {
    tnear = 0.0f;
  }

  // Calc hits with clip sphere.
  float tsnear;
  float tsfar;
  if (t_clipSphere)
  {
    // In probe mode, rays that don't hit the sphere simply aren't rendered.
    // In ordinary sphere mode, the intersection data is memorized.
    if (!intersectSphere(ray, sphereCenter, sphereRadius, &tsnear, &tsfar) && t_useSphereAsProbe)
    {
      d_output[y * texwidth + x] = make_uchar4(0);
      return;
    }
  }

  // Calc hits with clip plane.
  float tpnear;
  float nddot;
  if (t_clipPlane)
  {
    intersectPlane(ray, planeNormal, planeDist, &nddot, &tpnear);
  }

  float4 dst;

  if (t_mipMode > 0)
  {
    dst = backgroundColor;
  }
  else
  {
    dst = make_float4(0.0f);
  }

  float t = tnear;
  float3 pos = ray.o + ray.d * tnear;

  if (t_jittering)
  {
    const float4 randOffset = tex1D(randTexture, (y * width + x) % NUM_RAND_VECS);
    pos += make_float3(randOffset);
  }
  const float3 step = ray.d * dist;

  // If just clipped, shade with the normal of the clipping surface.
  bool justClippedPlane = false;
  bool justClippedSphere = false;

  float maxDiff = 0.;
  float3 maxDiffDepth = make_float3(0., 0., 0.);
  float lastAlpha = 0.;
  for (int i=0; i<maxSteps; ++i)
  {
    // Test for clipping.
    const bool clippedPlane = (t_clipPlane && (((t <= tpnear) && (nddot >= 0.0f))
                                            || ((t >= tpnear) && (nddot < 0.0f))));
    const bool clippedSphere = t_useSphereAsProbe ? (t_clipSphere && ((t < tsnear) || (t > tsfar)))
                                                  : (t_clipSphere && (t >= tsnear) && (t <= tsfar));

    if (clippedPlane || clippedSphere)
    {
      justClippedPlane = clippedPlane;
      justClippedSphere = clippedSphere;

      t += dist;
      if (t > tfar)
      {
        break;
      }
      pos += step;
      continue;
    }

    const float3 texCoord = calcTexCoord(pos, volPos, volSizeHalf);

    // Skip over homogeneous space.
    if (t_spaceSkipping)
    {
      if (skipSpace(texCoord))
      {
        t += dist;
        if (t > tfar)
        {
          break;
        }
        pos += step;
        continue;
      }
    }

    const float sample = volume<t_bpc>(texCoord);

    // Post-classification transfer-function lookup.
    float4 src = tex1D(tfTexture, sample);

    if (t_mipMode == 1)
    {
      dst.x = fmaxf(src.x, dst.x);
      dst.y = fmaxf(src.y, dst.y);
      dst.z = fmaxf(src.z, dst.z);
      dst.w = 1;
    }
    else if (t_mipMode == 2)
    {
      dst.x = fminf(src.x, dst.x);
      dst.y = fminf(src.y, dst.y);
      dst.z = fminf(src.z, dst.z);
      dst.w = 1;
    }

    // Local illumination.
    if (t_lighting && (src.w > 0.1f))
    {
      const float3 Ka = make_float3(0.0f, 0.0f, 0.0f);
      const float3 Kd = make_float3(0.8f, 0.8f, 0.8f);
      const float3 Ks = make_float3(0.8f, 0.8f, 0.8f);
      const float shininess = 1000.0f;
      if (justClippedPlane)
      {
        src = blinnPhong<t_bpc>(src, texCoord, L, H, Ka, Kd, Ks, shininess, &planeNormal);
        justClippedPlane = false;
      }
      else if (justClippedSphere)
      {
        float3 sphereNormal = normalize(pos - sphereCenter);
        src = blinnPhong<t_bpc>(src, texCoord, L, H, Ka, Kd, Ks, shininess, &sphereNormal);
        justClippedSphere = false;
      }
      else
      {
        src = blinnPhong<t_bpc>(src, texCoord, L, H, Ka, Kd, Ks, shininess);
      }
    }
    justClippedPlane = false;
    justClippedSphere = false;

    if (t_opacityCorrection)
    {
      src.w = 1 - powf(1 - src.w, dist);
    }

    if (t_mipMode == 0)
    {
      // pre-multiply alpha
      src.x *= src.w;
      src.y *= src.w;
      src.z *= src.w;
    }

    if (t_frontToBack && (t_mipMode == 0))
    {
      dst = dst + src * (1.0f - dst.w);
    }
    else if (!t_frontToBack && (t_mipMode == 0))
    {
      //dst = dst * src.w + src * (1.0f - src.w);
    }

    if (t_earlyRayTermination && (dst.w > opacityThreshold))
    {
      break;
    }

    t += dist;
    if (t > tfar)
    {
      break;
    }

    pos += step;

    if(t_isaDepth)
    {
      if(dst.w - lastAlpha > maxDiff)
      {
        maxDiff = dst.w - lastAlpha;
        maxDiffDepth = pos;
      }
      lastAlpha = dst.w;
    }
  }
  if(t_isaDepth)
  {
    // convert position to window-coordinates
    const float4 depthWin = mulPost(c_MvPrMatrix, make_float4(maxDiffDepth.x, maxDiffDepth.y, maxDiffDepth.z, 1.0f));
    float3 depth = perspectiveDivide(depthWin);
    // map and clip on near and far-clipping planes
    depth.z++;
    depth.z = depth.z/2.;

    if(depth.z > 1.0)
      depth.z = 1.0;
    else if(depth.z < 0.0)
      depth.z = 0.0;

    switch(dp)
    {
    case vvImage2_5d::VV_UCHAR:
      ((unsigned char*)(d_depth))[y * texwidth + x] = (unsigned char)(depth.z*float(UCHAR_MAX));
      break;
    case vvImage2_5d::VV_USHORT:
      ((unsigned short*)(d_depth))[y * texwidth + x] = (unsigned short)(depth.z*float(USHRT_MAX));
      break;
    case vvImage2_5d::VV_UINT:
      default:
      ((unsigned int*)(d_depth))[y * texwidth + x] = (unsigned int)(depth.z*float(UINT_MAX));
      break;
    }
  }
  d_output[y * texwidth + x] = rgbaFloatToInt(dst);
}

typedef void(*renderKernel)(uchar4*, const uint, const uint, const float4,
                            const uint, const float, const float3, const float3,
                            const float3, const float3, const float3, const float3, const float3,
                            const float, const float3, const float, void*, vvImage2_5d::DepthPrecision);

template<
         int t_bpc,
         bool t_illumination,
         bool t_opacityCorrection,
         bool t_earlyRayTermination,
         bool t_clipPlane,
         bool t_clipSphere,
         bool t_useSphereAsProbe,
         int t_mipMode
        >
renderKernel getKernelWithMip(vvRayRend*)
{
  return &render<t_earlyRayTermination, // Early ray termination.
                 true, // Space skipping.
                 true, // Front to back.
                 t_bpc, // Bytes per channel.
                 t_mipMode, // Mip mode.
                 t_illumination, // Local illumination.
                 t_opacityCorrection, // Opacity correction.
                 false, // Jittering.
                 t_clipPlane, // Clip plane.
                 t_clipSphere, // Clip sphere.
                 t_useSphereAsProbe // Show what's inside the clip sphere.
                >;
}

#ifdef FAST_COMPILE
template<
         int t_bpc,
         bool t_illumination,
         bool t_opacityCorrection,
         bool t_earlyRayTermination,
         bool t_clipPlane,
         bool t_clipSphere,
         bool t_useSphereAsProbe,
         int t_mipMode
        >
renderKernel getKernel(vvRayRend*)
{
  return &render<t_earlyRayTermination, // Early ray termination.
                 true, // Space skipping.
                 true, // Front to back.
                 t_bpc, // Bytes per channel.
                 t_mipMode, // Mip mode.
                 t_illumination, // Local illumination.
                 t_opacityCorrection, // Opacity correction.
                 false, // Jittering.
                 t_clipPlane, // Clip plane.
                 t_clipSphere, // Clip sphere.
                 t_useSphereAsProbe // Show what's inside the clip sphere.
                >;
}
#else
template<
         int t_bpc,
         bool t_illumination,
         bool t_opacityCorrection,
         bool t_earlyRayTermination,
         bool t_clipPlane,
         bool t_clipSphere,
         bool t_useSphereAsProbe
        >
renderKernel getKernelWithSphereAsProbe(vvRayRend* rayRend)
{
  switch ((int)rayRend->getParameter(vvRenderState::VV_MIP_MODE))
  {
  case 0:
    return getKernelWithMip<
                            t_bpc,
                            t_illumination,
                            t_opacityCorrection,
                            t_earlyRayTermination,
                            t_clipPlane,
                            t_clipSphere,
                            t_useSphereAsProbe,
                            0
                           >(rayRend);
  case 1:
    // No early ray termination possible with max intensity projection.
    return getKernelWithMip<
                            t_bpc,
                            t_illumination,
                            t_opacityCorrection,
                            false,
                            t_clipPlane,
                            t_clipSphere,
                            t_useSphereAsProbe,
                            1
                           >(rayRend);
  case 2:
    // No early ray termination possible with min intensity projection.
    return getKernelWithMip<
                            t_bpc,
                            t_illumination,
                            t_opacityCorrection,
                            false,
                            t_clipPlane,
                            t_clipSphere,
                            t_useSphereAsProbe,
                            2
                           >(rayRend);
  default:
    return getKernelWithMip<
                            t_bpc,
                            t_illumination,
                            t_opacityCorrection,
                            t_earlyRayTermination,
                            t_clipPlane,
                            t_clipSphere,
                            t_useSphereAsProbe,
                            0
                           >(rayRend);
  }
}

template<
         int t_bpc,
         bool t_illumination,
         bool t_opacityCorrection,
         bool t_earlyRayTermination,
         bool t_clipPlane
        >
renderKernel getKernelWithClipPlane(vvRayRend* rayRend)
{
  if ((bool)rayRend->getParameter(vvRenderState::VV_IS_ROI_USED)
     && (bool)rayRend->getParameter(vvRenderState::VV_SPHERICAL_ROI))
  {
    return getKernelWithSphereAsProbe<
                                      t_bpc,
                                      t_illumination,
                                      t_opacityCorrection,
                                      t_earlyRayTermination,
                                      t_clipPlane,
                                      true,
                                      true
                                     >(rayRend);
  }
  else
  {
    return getKernelWithSphereAsProbe<
                                      t_bpc,
                                      t_illumination,
                                      t_opacityCorrection,
                                      t_earlyRayTermination,
                                      t_clipPlane,
                                      false,
                                      false
                                     >(rayRend);
  }
}

template<
         int t_bpc,
         bool t_illumination,
         bool t_opacityCorrection,
         bool t_earlyRayTermination
        >
renderKernel getKernelWithEarlyRayTermination(vvRayRend* rayRend)
{
  if (rayRend->getParameter(vvRenderState::VV_CLIP_MODE))
  {
    return getKernelWithClipPlane<
                                  t_bpc,
                                  t_illumination,
                                  t_opacityCorrection,
                                  t_earlyRayTermination,
                                  true
                                 >(rayRend);
  }
  else
  {
    {
      return getKernelWithClipPlane<
                                    t_bpc,
                                    t_illumination,
                                    t_opacityCorrection,
                                    t_earlyRayTermination,
                                    false
                                   >(rayRend);
    }
  }
}

template<
         int t_bpc,
         bool t_illumination,
         bool t_opacityCorrection
        >
renderKernel getKernelWithOpacityCorrection(vvRayRend* rayRend)
{
  if (rayRend->getEarlyRayTermination())
  {
    return getKernelWithEarlyRayTermination<
                                            t_bpc,
                                            t_illumination,
                                            t_opacityCorrection,
                                            true
                                           >(rayRend);
  }
  else
  {
    return getKernelWithEarlyRayTermination<
                                            t_bpc,
                                            t_illumination,
                                            t_opacityCorrection,
                                            false
                                           >(rayRend);
  }
}

template<
         int t_bpc,
         bool t_illumination
        >
renderKernel getKernelWithIllumination(vvRayRend* rayRend)
{
  if (rayRend->getOpacityCorrection())
  {
    return getKernelWithOpacityCorrection<t_bpc, t_illumination, true>(rayRend);
  }
  else
  {
    return getKernelWithOpacityCorrection<t_bpc, t_illumination, false>(rayRend);
  }
}

template<
         int t_bpc
        >
renderKernel getKernelWithBpc(vvRayRend* rayRend)
{
  if (rayRend->getIllumination())
  {
    return getKernelWithIllumination<t_bpc, true>(rayRend);
  }
  else
  {
    return getKernelWithIllumination<t_bpc, false>(rayRend);
  }
}

renderKernel getKernel(vvRayRend* rayRend)
{
  if (rayRend->getVolDesc()->bpc == 1)
  {
    return getKernelWithBpc<1>(rayRend);
  }
  else if (rayRend->getVolDesc()->bpc == 2)
  {
    return getKernelWithBpc<2>(rayRend);
  }
  else
  {
    return getKernelWithBpc<1>(rayRend);
  }
}
#endif

vvRayRend::vvRayRend(vvVolDesc* vd, vvRenderState renderState)
  : vvSoftVR(vd, renderState)
{
  glewInit();

  _volumeCopyToGpuOk = true;

  _earlyRayTermination = true;
  _illumination = false;
  _interpolation = true;
  _opacityCorrection = true;
  _spaceSkipping = false;
  h_spaceSkippingArray = NULL;
  h_cellMinValues = NULL;
  h_cellMaxValues = NULL;

  _rgbaTF = NULL;

#if 0
  const int numCells[] = { 16, 16, 16 };
  setNumSpaceSkippingCells(numCells);
#endif
  d_spaceSkippingArray = 0;

  intImg = new vvCudaImg(0, 0);

  const vvCudaImg::Mode mode = dynamic_cast<vvCudaImg*>(intImg)->getMode();
  if (mode == vvCudaImg::TEXTURE)
  {
    setWarpMode(CUDATEXTURE);
  }

  factorViewMatrix();
  bool ignoreMe;
  vvCuda::checkError(&ignoreMe, cudaGetLastError(), "rayRend-constructor");
  d_randArray = 0;
  initRandTexture();

  initVolumeTexture();

  d_transferFuncArray = 0;
  updateTransferFunction();
}

vvRayRend::~vvRayRend()
{
  bool ok;
  for (int f=0; f<d_volumeArrays.size(); ++f)
  {
    vvCuda::checkError(&ok, cudaFreeArray(d_volumeArrays[f]),
                       "vvRayRend::~vvRayRend() - free volume frame");
  }

  vvCuda::checkError(&ok, cudaFreeArray(d_transferFuncArray),
                     "vvRayRend::~vvRayRend() - free tf");
  vvCuda::checkError(&ok, cudaFreeArray(d_randArray),
                     "vvRayRend::~vvRayRend() - free rand array");
  vvCuda::checkError(&ok, cudaFreeArray(d_spaceSkippingArray),
                     "vvRayRend::~vvRayRend() - free space skipping array");

  delete[] h_spaceSkippingArray;
  delete[] h_cellMinValues;
  delete[] h_cellMaxValues;
  delete[] _rgbaTF;
}

int vvRayRend::getLUTSize() const
{
   vvDebugMsg::msg(2, "vvRayRend::getLUTSize()");
   return (vd->getBPV()==2) ? 4096 : 256;
}

void vvRayRend::updateTransferFunction()
{
  bool ok;

  int lutEntries = getLUTSize();
  delete[] _rgbaTF;
  _rgbaTF = new float[4 * lutEntries];

  vd->computeTFTexture(lutEntries, 1, 1, _rgbaTF);

  if (_spaceSkipping)
  {
    computeSpaceSkippingTexture();
    initSpaceSkippingTexture();
  }

  cudaChannelFormatDesc channelDesc = cudaCreateChannelDesc<float4>();

  vvCuda::checkError(&ok, cudaFreeArray(d_transferFuncArray),
                     "vvRayRend::updateTransferFunction() - free tf texture");
  vvCuda::checkError(&ok, cudaMallocArray(&d_transferFuncArray, &channelDesc, lutEntries, 1),
                     "vvRayRend::updateTransferFunction() - malloc tf texture");
  vvCuda::checkError(&ok, cudaMemcpyToArray(d_transferFuncArray, 0, 0, _rgbaTF, lutEntries * 4 * sizeof(float),
                                            cudaMemcpyHostToDevice),
                     "vvRayRend::updateTransferFunction() - copy tf texture to device");


  tfTexture.filterMode = cudaFilterModeLinear;
  tfTexture.normalized = true;    // access with normalized texture coordinates
  tfTexture.addressMode[0] = cudaAddressModeClamp;   // wrap texture coordinates

  vvCuda::checkError(&ok, cudaBindTextureToArray(tfTexture, d_transferFuncArray, channelDesc),
                     "vvRayRend::updateTransferFunction() - bind tf texture");
}

void vvRayRend::compositeVolume(int w, int h)
{
  if(!_volumeCopyToGpuOk)
  {
    std::cerr << "vvRayRend::compositeVolume() aborted because of previous CUDA-Error" << std::endl;
    return;
  }
  vvDebugMsg::msg(1, "vvRayRend::compositeVolume()");

  vvGLTools::Viewport vp = vvGLTools::getViewport();

  if ((w > 0) && (h > 0))
  {
    vp[2]=w; vp[3]=h;
    intImg->setSize(w, h);

    switch(_depthPrecision)
    {
    case vvImage2_5d::VV_UCHAR:
      cudaMalloc(&_depthUchar, w*h*sizeof(unsigned char));
      break;
    case vvImage2_5d::VV_USHORT:
      cudaMalloc(&_depthUshort, w*h*sizeof(unsigned short));
      break;
    case vvImage2_5d::VV_UINT:
      cudaMalloc(&_depthUint, w*h*sizeof(unsigned int));
      break;
    }
  }

  vp.print();

  dynamic_cast<vvCudaImg*>(intImg)->map();

  dim3 blockSize(16, 16);
  dim3 gridSize = dim3(iDivUp(vp[2], blockSize.x), iDivUp(vp[3], blockSize.y));
  const vvVector3 size(vd->getSize());

  vvVector3 probePosObj;
  vvVector3 probeSizeObj;
  vvVector3 probeMin;
  vvVector3 probeMax;
  calcProbeDims(probePosObj, probeSizeObj, probeMin, probeMax);

  vvVector3 clippedProbeSizeObj;
  clippedProbeSizeObj.copy(&probeSizeObj);
  for (int i=0; i<3; ++i)
  {
    if (clippedProbeSizeObj[i] < vd->getSize()[i])
    {
      clippedProbeSizeObj[i] = vd->getSize()[i];
    }
  }

  if (_isROIUsed && !_sphericalROI)
  {
    drawBoundingBox(&probeSizeObj, &_roiPos, &_probeColor);
  }

  const float diagonalVoxels = sqrtf(float(vd->vox[0] * vd->vox[0] +
                                           vd->vox[1] * vd->vox[1] +
                                           vd->vox[2] * vd->vox[2]));
  int numSlices = max(1, static_cast<int>(_quality * diagonalVoxels));

  vvMatrix Mv, MvPr;
  getModelviewMatrix(&Mv);
  getProjectionMatrix(&MvPr);
  MvPr.multiplyPre(&Mv);

  float* mvprM = new float[16];
  MvPr.get(mvprM);
  cudaMemcpyToSymbol(c_MvPrMatrix, mvprM, sizeof(float4) * 4);

  vvMatrix invMv;
  invMv.copy(&Mv);
  invMv.invert();

  vvMatrix pr;
  getProjectionMatrix(&pr);

  vvMatrix invMvpr;
  getModelviewMatrix(&invMvpr);
  invMvpr.multiplyPost(&pr);
  invMvpr.invert();

  float* viewM = new float[16];
  invMvpr.get(viewM);
  cudaMemcpyToSymbol(c_invViewMatrix, viewM, sizeof(float4) * 4);
  delete[] viewM;

  const float3 volPos = make_float3(vd->pos[0], vd->pos[1], vd->pos[2]);
  float3 probePos = volPos;
  if (_isROIUsed && !_sphericalROI)
  {
    probePos = make_float3(probePosObj[0],  probePosObj[1], probePosObj[2]);
  }
  vvVector3 sz = vd->getSize();
  const float3 volSize = make_float3(sz[0], sz[1], sz[2]);
  float3 probeSize = make_float3(probeSizeObj[0], probeSizeObj[1], probeSizeObj[2]);
  if (_sphericalROI)
  {
    probeSize = make_float3(vd->vox[0], vd->vox[1], vd->vox[2]);
  }

  const bool isOrtho = pr.isProjOrtho();

  vvVector3 eye;
  getEyePosition(&eye);
  eye.multiply(&invMv);

  vvVector3 origin;

  vvVector3 normal;
  getShadingNormal(normal, origin, eye, invMv, isOrtho);

  const float3 N = make_float3(normal[0], normal[1], normal[2]);

  const float3 L(-N);

  // Viewing direction.
  const float3 V(-N);

  // Half way vector.
  const float3 H = normalize(L + V);

  // Clip sphere.
  const float3 center = make_float3(_roiPos[0], _roiPos[1], _roiPos[2]);
  const float radius = _roiSize[0] * vd->getSize()[0];

  // Clip plane.
  const float3 pnormal = normalize(make_float3(_clipNormal[0], _clipNormal[1], _clipNormal[2]));
  const float pdist = _clipNormal.dot(&_clipPoint);

  if (_clipMode && _clipPerimeter)
  {
    drawPlanePerimeter(&size, &vd->pos, &_clipPoint, &_clipNormal, &_clipColor);
  }

  GLfloat bgcolor[4];
  glGetFloatv(GL_COLOR_CLEAR_VALUE, bgcolor);
  float4 backgroundColor = make_float4(bgcolor[0], bgcolor[1], bgcolor[2], bgcolor[3]);

#ifdef FAST_COMPILE
  renderKernel kernel = getKernel<
                        1,
                        true, // Local illumination.
                        true, // Opacity correction
                        true, // Early ray termination.
                        false, // Use clip plane.
                        false, // Use clip sphere.
                        false, // Use clip sphere as probe (inverted sphere).
                        0 // Mip mode.
                       >(this);
#else
  renderKernel kernel = getKernel(this);
#endif

  if (kernel != NULL)
  {
    if (vd->bpc == 1)
    {
      cudaBindTextureToArray(volTexture8, d_volumeArrays[vd->getCurrentFrame()], _channelDesc);
    }
    else if (vd->bpc == 2)
    {
      cudaBindTextureToArray(volTexture16, d_volumeArrays[vd->getCurrentFrame()], _channelDesc);
    }
    switch(_depthPrecision)
    {
    case vvImage2_5d::VV_UCHAR:
      (kernel)<<<gridSize, blockSize>>>(dynamic_cast<vvCudaImg*>(intImg)->getDImg(), vp[2], vp[3],
                                        backgroundColor, intImg->width,diagonalVoxels / (float)numSlices,
                                        volPos, volSize * 0.5f,
                                        probePos, probeSize * 0.5f,
                                        L, H,
                                        center, radius * radius,
                                        pnormal, pdist, _depthUchar, _depthPrecision);
      break;
    case vvImage2_5d::VV_USHORT:
      (kernel)<<<gridSize, blockSize>>>(dynamic_cast<vvCudaImg*>(intImg)->getDImg(), vp[2], vp[3],
                                        backgroundColor, intImg->width,diagonalVoxels / (float)numSlices,
                                        volPos, volSize * 0.5f,
                                        probePos, probeSize * 0.5f,
                                        L, H,
                                        center, radius * radius,
                                        pnormal, pdist, _depthUshort, _depthPrecision);
      break;
    case vvImage2_5d::VV_UINT:
      (kernel)<<<gridSize, blockSize>>>(dynamic_cast<vvCudaImg*>(intImg)->getDImg(), vp[2], vp[3],
                                        backgroundColor, intImg->width,diagonalVoxels / (float)numSlices,
                                        volPos, volSize * 0.5f,
                                        probePos, probeSize * 0.5f,
                                        L, H,
                                        center, radius * radius,
                                        pnormal, pdist, _depthUint, _depthPrecision);
      break;
    }
  }
  dynamic_cast<vvCudaImg*>(intImg)->unmap();

  // For bounding box, tf palette display, etc.
  vvRenderer::renderVolumeGL();
}
//----------------------------------------------------------------------------
// see parent
void vvRayRend::setParameter(const ParameterType param, const float newValue)
{
  vvDebugMsg::msg(3, "vvTexRend::setParameter()");

  switch (param)
  {
  case vvRenderer::VV_SLICEINT:
    {
      const bool newInterpol = static_cast<bool>(newValue);
      if (_interpolation != newInterpol)
      {
        _interpolation = newInterpol;
        initVolumeTexture();
        updateTransferFunction();

        if (_spaceSkipping)
        {
          initSpaceSkippingTexture();
        }
      }
    }
    break;
  case vvRenderer::VV_LIGHTING:
    _illumination = static_cast<bool>(newValue);
    break;
  case vvRenderer::VV_OPCORR:
    _opacityCorrection = static_cast<bool>(newValue);
    break;
  case vvRenderer::VV_TERMINATEEARLY:
    _earlyRayTermination = static_cast<bool>(newValue);
    break;
  default:
    vvRenderer::setParameter(param, newValue);
    break;
  }
}

void vvRayRend::setNumSpaceSkippingCells(const int numSpaceSkippingCells[3])
{
  for (int d=0; d<3; ++d)
  {
    h_numCells[d] = numSpaceSkippingCells[d];
  }
  calcSpaceSkippingGrid();
}

bool vvRayRend::getEarlyRayTermination() const
{
  return _earlyRayTermination;
}
bool vvRayRend::getIllumination() const
{
  return _illumination;
}

bool vvRayRend::getInterpolation() const
{
  return _interpolation;
}

bool vvRayRend::getOpacityCorrection() const
{
  return _opacityCorrection;
}

bool vvRayRend::getSpaceSkipping() const
{
  return _spaceSkipping;
}

void vvRayRend::initRandTexture()
{
  bool ok;

  const float scale = 2.0f;

  //srand(time(NULL));

  float4* randVecs = new float4[NUM_RAND_VECS];
  for (int i=0; i<NUM_RAND_VECS; ++i)
  {
    randVecs[i].x = (static_cast<float>(rand()) / static_cast<float>(INT_MAX)) * scale;
    randVecs[i].y = (static_cast<float>(rand()) / static_cast<float>(INT_MAX)) * scale;
    randVecs[i].z = (static_cast<float>(rand()) / static_cast<float>(INT_MAX)) * scale;
  }

  cudaChannelFormatDesc channelDesc = cudaCreateChannelDesc<float4>();

  vvCuda::checkError(&ok, cudaFreeArray(d_randArray),
                     "vvRayRend::initRandTexture()");
  vvCuda::checkError(&ok, cudaMallocArray(&d_randArray, &channelDesc, NUM_RAND_VECS, 1),
                     "vvRayRend::initRandTexture()");
  vvCuda::checkError(&ok, cudaMemcpyToArray(d_randArray, 0, 0, randVecs, NUM_RAND_VECS * sizeof(float4),
                                            cudaMemcpyHostToDevice), "vvRayRend::initRandTexture()");

  randTexture.filterMode = cudaFilterModeLinear;
  randTexture.addressMode[0] = cudaAddressModeClamp;

  vvCuda::checkError(&ok, cudaBindTextureToArray(randTexture, d_randArray, channelDesc),
                     "vvRayRend::initRandTexture()");

  delete[] randVecs;
}

void vvRayRend::initSpaceSkippingTexture()
{
  bool ok;

  cudaExtent numBricks = make_cudaExtent(h_numCells[0], h_numCells[1], h_numCells[2]);

  cudaChannelFormatDesc channelDesc = cudaCreateChannelDesc<bool>();

  vvCuda::checkError(&ok, cudaFreeArray(d_spaceSkippingArray),
                     "vvRayRend::initSpaceSkippingTexture()");
  vvCuda::checkError(&ok, cudaMalloc3DArray(&d_spaceSkippingArray, &channelDesc, numBricks),
                     "vvRayRend::initSpaceSkippingTexture()");

  cudaMemcpy3DParms copyParams = { 0 };

  copyParams.srcPtr = make_cudaPitchedPtr(h_spaceSkippingArray, numBricks.width*vd->bpc, numBricks.width, numBricks.height);
  copyParams.dstArray = d_spaceSkippingArray;
  copyParams.extent = numBricks;
  copyParams.kind = cudaMemcpyHostToDevice;
  vvCuda::checkError(&ok, cudaMemcpy3D(&copyParams), "vvRayRend::initSpaceSkippingTexture()");
}

void vvRayRend::initVolumeTexture()
{
  bool ok;

  cudaExtent volumeSize = make_cudaExtent(vd->vox[0], vd->vox[1], vd->vox[2]);
  if (vd->bpc == 1)
  {
    _channelDesc = cudaCreateChannelDesc<uchar>();
  }
  else if (vd->bpc == 2)
  {
    _channelDesc = cudaCreateChannelDesc<ushort>();
  }
  d_volumeArrays.resize(vd->frames);

  // Free "cuda error cache".
  vvCuda::checkError(&ok, cudaGetLastError(), "vvRayRend::initVolumeTexture() - free cuda error cache");

  int outOfMemFrame = -1;
  for (int f=0; f<vd->frames; ++f)
  {
    vvCuda::checkError(&_volumeCopyToGpuOk, cudaMalloc3DArray(&d_volumeArrays[f],
                                            &_channelDesc,
                                            volumeSize),
                       "vvRayRend::initVolumeTexture() - try to alloc 3D array");
    size_t availableMem;
    size_t totalMem;
    vvCuda::checkError(&ok, cudaMemGetInfo(&availableMem, &totalMem),
                       "vvRayRend::initVolumeTexture() - get mem info from device");

    if(!_volumeCopyToGpuOk)
    {
      outOfMemFrame = f;
      break;
    }

    vvDebugMsg::msg(1, "Total CUDA memory:     ", (int)totalMem);
    vvDebugMsg::msg(1, "Available CUDA memory: ", (int)availableMem);

    cudaMemcpy3DParms copyParams = { 0 };

    if (vd->bpc == 1)
    {
      copyParams.srcPtr = make_cudaPitchedPtr(vd->getRaw(f), volumeSize.width*vd->bpc, volumeSize.width, volumeSize.height);
    }
    else if (vd->bpc == 2)
    {
      const int size = vd->vox[0] * vd->vox[1] * vd->vox[2] * vd->bpc;
      uchar* raw = vd->getRaw(0);
      uchar* data = new uchar[size];

      for (int i=0; i<size; i+=2)
      {
        int val = ((int) raw[i] << 8) | (int) raw[i + 1];
        val >>= 4;
        data[i] = raw[i];
        data[i + 1] = val;
      }
      copyParams.srcPtr = make_cudaPitchedPtr(data, volumeSize.width*vd->bpc, volumeSize.width, volumeSize.height);
    }
    copyParams.dstArray = d_volumeArrays[f];
    copyParams.extent = volumeSize;
    copyParams.kind = cudaMemcpyHostToDevice;
    vvCuda::checkError(&ok, cudaMemcpy3D(&copyParams),
                       "vvRayRend::initVolumeTexture() - copy volume frame to 3D array");
  }

  if (outOfMemFrame >= 0)
  {
    cerr << "Couldn't accomodate the volume" << endl;
    for (int f=0; f<=outOfMemFrame; ++f)
    {
      vvCuda::checkError(&ok, cudaFree(d_volumeArrays[f]),
                         "vvRayRend::initVolumeTexture() - free memory after failure");
      d_volumeArrays[f] = NULL;
    }
  }

  if (vd->bpc == 1)
  {
    for (int f=0; f<outOfMemFrame; ++f)
    {
      vvCuda::checkError(&ok, cudaFreeArray(d_volumeArrays[f]),
                         "vvRayRend::initVolumeTexture() - why do we do this right here?");
      d_volumeArrays[f] = NULL;
    }
  }

  if (_volumeCopyToGpuOk)
  {
    if (vd->bpc == 1)
    {
        volTexture8.normalized = true;
        if (_interpolation)
        {
          volTexture8.filterMode = cudaFilterModeLinear;
        }
        else
        {
          volTexture8.filterMode = cudaFilterModePoint;
        }
        volTexture8.addressMode[0] = cudaAddressModeClamp;
        volTexture8.addressMode[1] = cudaAddressModeClamp;
        vvCuda::checkError(&ok, cudaBindTextureToArray(volTexture8, d_volumeArrays[0], _channelDesc),
                           "vvRayRend::initVolumeTexture() - bind volume texture (bpc == 1)");
    }
    else if (vd->bpc == 2)
    {
        volTexture16.normalized = true;
        if (_interpolation)
        {
          volTexture16.filterMode = cudaFilterModeLinear;
        }
        else
        {
          volTexture16.filterMode = cudaFilterModePoint;
        }
        volTexture16.addressMode[0] = cudaAddressModeClamp;
        volTexture16.addressMode[1] = cudaAddressModeClamp;
        vvCuda::checkError(&ok, cudaBindTextureToArray(volTexture16, d_volumeArrays[0], _channelDesc),
                           "vvRayRend::initVolumeTexture() - bind volume texture (bpc == 2)");
    }
  }
}

void vvRayRend::factorViewMatrix()
{
  vvGLTools::Viewport vp = vvGLTools::getViewport();
  const int w = vvToolshed::getTextureSize(vp[2]);
  const int h = vvToolshed::getTextureSize(vp[3]);

  if ((intImg->width != w) || (intImg->height != h))
  {
    intImg->setSize(w, h);
  }

  iwWarp.identity();
  iwWarp.translate(-1.0f, -1.0f, 0.0f);
  iwWarp.scale(1.0f / (static_cast<float>(vp[2]) * 0.5f), 1.0f / (static_cast<float>(vp[3]) * 0.5f), 0.0f);
}

void vvRayRend::findAxisRepresentations()
{
  // Overwrite default behavior.
}

void vvRayRend::calcSpaceSkippingGrid()
{
  delete[] h_cellMinValues;
  delete[] h_cellMaxValues;
  const int numCells = h_numCells[0] * h_numCells[1] * h_numCells[2];
  h_cellMinValues = new int[numCells];
  h_cellMaxValues = new int[numCells];

  // Cells are uniformly sized. If vd->size[d] isn't a multiple of cellSize[d],
  // the last cell will be larger than the other cells for that dimension.
  const int cellSizeAll[] = {
                              (int)vd->getSize()[0] / (h_numCells[0]),
                              (int)vd->getSize()[1] / (h_numCells[1]),
                              (int)vd->getSize()[2] / (h_numCells[2])
                          };
  const int lastCellSize[] = {
                               cellSizeAll[0] + (int)vd->getSize()[0] % h_numCells[0],
                               cellSizeAll[1] + (int)vd->getSize()[1] % h_numCells[1],
                               cellSizeAll[2] + (int)vd->getSize()[2] % h_numCells[2]
                              };
  int cellSize[3];
  int from[3];
  int to[3];
  const vvVector3 size = vd->getSize();
  for (int z=0; z<h_numCells[2]; ++z)
  {
    cellSize[2] = (z == (h_numCells[2] - 1)) ? lastCellSize[2] : cellSizeAll[2];
    from[2] = cellSizeAll[2] * z;
    to[2] = from[2] + cellSize[2];

    for (int y=0; y<h_numCells[1]; ++y)
    {
      cellSize[1] = (y == (h_numCells[1] - 1)) ? lastCellSize[1] : cellSizeAll[1];
      from[1] = cellSizeAll[1] * y;
      to[1] = from[1] + cellSize[1];

      for (int x=0; x<h_numCells[2]; ++x)
      {
        cellSize[0] = (x == (h_numCells[0] - 1)) ? lastCellSize[0] : cellSizeAll[0];
        from[0] = cellSizeAll[0] * x;
        to[0] = from[0] + cellSize[0];

        // Memorize the max and min scalar values in the volume. These are stored
        // to perform space skipping later on.
        int minValue = INT_MAX;
        int maxValue = -INT_MAX;

        for (int vz=from[2]; vz<to[2]; ++vz)
        {
          for (int vy=from[1]; vy<to[1]; ++vy)
          {
            for (int vx=from[0]; vx<to[0]; ++vx)
            {
              const uchar vox = vd->getRaw()[vz * (int)(size[0] * size[1]) + vy * (int)size[0] + vx];

              // Store min and max for empty space leaping.
              if (vox > maxValue)
              {
                maxValue = vox;
              }
              if (vox < minValue)
              {
                minValue = vox;
              }
            }
          }
        }
        const int idx = x * h_numCells[1] * h_numCells[2] + y * h_numCells[2] + z;
        h_cellMinValues[idx] = minValue;
        h_cellMaxValues[idx] = maxValue;
      }
    }
  }
}

void vvRayRend::computeSpaceSkippingTexture()
{
  if (vd->bpc == 1)
  {
    delete[] h_spaceSkippingArray;
    const int numCells = h_numCells[0] * h_numCells[1] * h_numCells[2];
    h_spaceSkippingArray = new bool[numCells];
    int discarded = 0;
    for (int i=0; i<numCells; ++i)
    {
      h_spaceSkippingArray[i] = true;
      for (int j=h_cellMinValues[i]; j<=h_cellMaxValues[i]; ++j)
      {
        if(_rgbaTF[j * 4 + 3] > 0.0f)
        {
          h_spaceSkippingArray[i] = false;
          break;
        }
      }
      if (h_spaceSkippingArray[i])
      {
        ++discarded;
      }
    }
    vvDebugMsg::msg(3, "Cells discarded: ", discarded);
  }
  else
  {
    // Only for bpc == 1 so far.
    _spaceSkipping = false;
    delete[] h_spaceSkippingArray;
    h_spaceSkippingArray = NULL;
    delete[] h_cellMinValues;
    h_cellMinValues = NULL;
    delete[] h_cellMaxValues;
    h_cellMaxValues = NULL;
  }
}

void vvRayRend::setDepthPrecision(vvImage2_5d::DepthPrecision dp)
{
  _depthPrecision = dp;
}

#endif
