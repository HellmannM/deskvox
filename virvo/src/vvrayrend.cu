
#include <hip/hip_runtime.h>
//
// This software contains source code provided by NVIDIA Corporation.
//

#ifdef HAVE_CONFIG_H
#include "vvconfig.h"
#endif

#if defined(HAVE_CUDA) && defined(NV_PROPRIETARY_CODE)

#include "vvcudautils.h"
#include "vvdebugmsg.h"
#include "vvglew.h"
#include "vvgltools.h"
#include "vvrayrend.h"

#include <cuda.h>
#include <cuda_runtime_api.h>
#include <cuda_gl_interop.h>
#include <ctime>
#include <iostream>

using std::cerr;
using std::endl;

texture<uchar, 3, cudaReadModeNormalizedFloat> volTexture8;
texture<ushort, 3, cudaReadModeNormalizedFloat> volTexture16;
texture<float4, 1, cudaReadModeElementType> tfTexture;
texture<float4, 1, cudaReadModeElementType> randTexture;

const int NUM_RAND_VECS = 8192;

int iDivUp(const int a, const int b)
{
  return (a % b != 0) ? (a / b + 1) : (a / b);
}

typedef struct
{
  float m[4][4];
} matrix4x4;

__constant__ matrix4x4 c_invViewMatrix;

struct Ray
{
  float3 o;
  float3 d;
};

template<int bpc>
__device__ float volume(const float x, const float y, const float z)
{
  if (bpc == 1)
  {
    return tex3D(volTexture8, x, y, z);
  }
  else if (bpc == 2)
  {
    return tex3D(volTexture16, x, y, z);
  }
  else
  {
    return -1.0f;
  }
}

template<int bpc>
__device__ float volume(const float3& pos)
{
  if (bpc == 1)
  {
    return tex3D(volTexture8, pos.x, pos.y, pos.z);
  }
  else if (bpc == 2)
  {
    return tex3D(volTexture16, pos.x, pos.y, pos.z);
  }
  else
  {
    return -1.0f;
  }
}

__device__ float3 calcTexCoord(const float3& pos, const float3& volSizeHalf)
{
  return make_float3((pos.x + volSizeHalf.x) / (volSizeHalf.x * 2.0f),
                     (pos.y + volSizeHalf.y) / (volSizeHalf.y * 2.0f),
                     (pos.z + volSizeHalf.z) / (volSizeHalf.z * 2.0f));
}

__device__ bool solveQuadraticEquation(const float A, const float B, const float C,
                                       float* tnear, float* tfar)
{
  const float discrim = B * B - 4.0f * A * C;
  if (discrim < 0.0f)
  {
    *tnear = -1.0f;
    *tfar = -1.0f;
  }
  const float rootDiscrim = __fsqrt_rn(discrim);
  float q;
  if (B < 0)
  {
    q = -0.5f * (B - rootDiscrim);
  }
  else
  {
    q = -0.5f * (B + rootDiscrim);
  }
  *tnear = q / A;
  *tfar = C / q;
  if (*tnear > *tfar)
  {
    float tmp = *tnear;
    *tnear = *tfar;
    *tfar = tmp;
    return true;
  }
  return false;
}

__device__ bool intersectBox(const Ray& ray, const float3& boxmin, const float3& boxmax,
                             float* tnear, float* tfar)
{
  // compute intersection of ray with all six bbox planes
  float3 invR = make_float3(1.0f, 1.0f, 1.0f) / ray.d;
  float t1 = (boxmin.x - ray.o.x) * invR.x;
  float t2 = (boxmax.x - ray.o.x) * invR.x;
  float tmin = fminf(t1, t2);
  float tmax = fmaxf(t1, t2);

  t1 = (boxmin.y - ray.o.y) * invR.y;
  t2 = (boxmax.y - ray.o.y) * invR.y;
  tmin = fmaxf(fminf(t1, t2), tmin);
  tmax = fminf(fmaxf(t1, t2), tmax);

  t1 = (boxmin.z - ray.o.z) * invR.z;
  t2 = (boxmax.z - ray.o.z) * invR.z;
  tmin = fmaxf(fminf(t1, t2), tmin);
  tmax = fminf(fmaxf(t1, t2), tmax);

  *tnear = tmin;
  *tfar = tmax;

  return ((tmax >= tmin) && (tmax >= 0.0f));
}

__device__ bool intersectSphere(const Ray& ray, const float3& center, const float radiusSqr,
                                float* tnear, float* tfar)
{
  Ray r = ray;
  r.o -= center;
  float A = r.d.x * r.d.x + r.d.y * r.d.y
          + r.d.z * r.d.z;
  float B = 2 * (r.d.x * r.o.x + r.d.y * r.o.y
               + r.d.z * r.o.z);
  float C = r.o.x * r.o.x + r.o.y * r.o.y
          + r.o.z * r.o.z - radiusSqr;
  return solveQuadraticEquation(A, B, C, tnear, tfar);
}

__device__ void intersectPlane(const Ray& ray, const float3& normal, const float& dist,
                               float* nddot, float* tnear)
{
  *nddot = dot(normal, ray.d);
  const float vOrigin = dist - dot(normal, ray.o);
  *tnear = vOrigin / *nddot;
}


__device__ float4 mul(const matrix4x4& M, const float4& v)
{
  float4 result;
  result.x = M.m[0][0] * v.x + M.m[0][1] * v.y + M.m[0][2] * v.z + M.m[0][3] * v.w;
  result.y = M.m[1][0] * v.x + M.m[1][1] * v.y + M.m[1][2] * v.z + M.m[1][3] * v.w;
  result.z = M.m[2][0] * v.x + M.m[2][1] * v.y + M.m[2][2] * v.z + M.m[2][3] * v.w;
  result.w = M.m[3][0] * v.x + M.m[3][1] * v.y + M.m[3][2] * v.z + M.m[3][3] * v.w;
  return result;
}

__device__ float3 perspectiveDivide(const float4& v)
{
  const float wInv = 1.0f / v.w;
  return make_float3(v.x * wInv, v.y * wInv, v.z * wInv);
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
  clamp(rgba.x);
  clamp(rgba.y);
  clamp(rgba.z);
  clamp(rgba.w);
  return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

__device__ uint rgbaFloatToInt(float3 rgb)
{
  float4 rgba = make_float4(rgb.x, rgb.y, rgb.z, 1.0f);
  return rgbaFloatToInt(rgba);
}

template<int bpc>
__device__ float3 gradient(const float3& pos)
{
  const float DELTA = 0.01f;

  float3 sample1;
  float3 sample2;

  sample1.x = volume<bpc>(pos - make_float3(DELTA, 0.0f, 0.0f));
  sample2.x = volume<bpc>(pos + make_float3(DELTA, 0.0f, 0.0f));
  sample1.y = volume<bpc>(pos - make_float3(0.0f, DELTA, 0.0f));
  sample2.y = volume<bpc>(pos + make_float3(0.0f, DELTA, 0.0f));
  sample1.z = volume<bpc>(pos - make_float3(0.0f, 0.0f, DELTA));
  sample2.z = volume<bpc>(pos + make_float3(0.0f, 0.0f, DELTA));

  return sample2 - sample1;
}

template<int bpc>
__device__ float4 blinnPhong(const float4& classification, const float3& pos,
                             const float3& L, const float3& H,
                             const float3& Ka, const float3& Kd, const float3& Ks,
                             const float shininess,
                             const float3* normal = NULL)
{
  float3 N = normalize(gradient<bpc>(pos));

  if (normal != NULL)
  {
    // Interpolate gradient with normal from clip object (based on opacity).
    N = (*normal * classification.w) + (N * (1.0f - classification.w));
    N = normalize(N);
  }

  const float diffuse = fabsf(dot(L, N));
  const float specular = powf(dot(H, N), shininess);

  const float3 c = make_float3(classification);
  float3 tmp = Ka * c + Kd * diffuse * c;
  if (specular > 0.0f)
  {
    tmp += Ks * specular * c;
  }
  return make_float4(tmp.x, tmp.y, tmp.z, classification.w);
}

template<
         bool earlyRayTermination,
         bool frontToBack,
         int bpc,
         int mipMode,
         bool lighting,
         bool opacityCorrection,
         bool jittering,
         bool clipSphere,
         bool clipPlane,
         bool useSphereAsProbe
        >
__global__ void render(uint *d_output, const uint width, const uint height, const float dist,
                       const float3 volSizeHalf, const float3 L, const float3 H,
                       const float3 sphereCenter, const float sphereRadius,
                       const float3 planeNormal, const float planeDist)
{
  const int maxSteps = INT_MAX;
  const float tstep = dist;
  const float opacityThreshold = 0.95f;

  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((x >= width) || (y >= height))
  {
    return;
  }

  const float u = (x / static_cast<float>(width)) * 2.0f - 1.0f;
  const float v = (y / static_cast<float>(height)) * 2.0f - 1.0f;

  /*
   * Rays like if the view were orthographic with origins over each pixel.
   * These are multiplied with the inverse modelview projection matrix.
   * First of all, the rays will be transformed to fit to the frustum.
   * Then the rays will be oriented so that they can hit the volume.
   */
  const float4 o = mul(c_invViewMatrix, make_float4(u, v, -1.0f, 1.0f));
  const float4 d = mul(c_invViewMatrix, make_float4(u, v, 1.0f, 1.0f));

  Ray ray;
  ray.o = perspectiveDivide(o);
  ray.d = perspectiveDivide(d);
  ray.d = ray.d - ray.o;
  ray.d = normalize(ray.d);

  float tnear;
  float tfar;
  const bool hit = intersectBox(ray, -volSizeHalf, volSizeHalf, &tnear, &tfar);
  if (!hit)
  {
    d_output[y * width + x] = 0;
    return;
  }

  if (tnear < 0.0f)
  {
    tnear = 0.0f;
  }

  // Calc hits with clip sphere.
  float tsnear;
  float tsfar;
  if (clipSphere)
  {
    // In probe mode, rays that don't hit the sphere simply aren't rendered.
    // In ordinary sphere mode, the intersection data is memorized.
    if (!intersectSphere(ray, sphereCenter, sphereRadius, &tsnear, &tsfar) && useSphereAsProbe)
    {
      d_output[y * width + x] = 0;
      return;
    }
  }

  // Calc hits with clip plane.
  float tpnear;
  float nddot;
  if (clipPlane)
  {
    intersectPlane(ray, planeNormal, planeDist, &nddot, &tpnear);
  }

  float4 dst = make_float4(0.0f);
  float t = tnear;
  float3 pos = ray.o + ray.d * tnear;

  if (jittering)
  {
    const float4 randOffset = tex1D(randTexture, (y * width + x) % NUM_RAND_VECS);
    pos += make_float3(randOffset);
  }
  const float3 step = ray.d * tstep;

  float maxIntensity = 0.0f;
  float minIntensity = FLT_MAX;

  // If just clipped, shade with the normal of the clipping surface.
  bool justClippedPlane = false;
  bool justClippedSphere = false;

  for (int i=0; i<maxSteps; ++i)
  {
    // Test for clipping.
    const bool clippedPlane = (clipPlane && (((t <= tpnear) && (nddot >= 0.0f))
                                          || ((t >= tpnear) && (nddot < 0.0f))));
    const bool clippedSphere = useSphereAsProbe ? (clipSphere && ((t < tsnear) || (t > tsfar)))
                                                : (clipSphere && (t >= tsnear) && (t <= tsfar));

    if (clippedPlane || clippedSphere)
    {
      justClippedPlane = clippedPlane;
      justClippedSphere = clippedSphere;

      t += tstep;
      if (t > tfar)
      {
        break;
      }
      pos += step;
      continue;
    }

    float3 texCoord = calcTexCoord(pos, volSizeHalf);

    const float sample = volume<bpc>(texCoord);

    // Post-classification transfer-function lookup.
    float4 src;

    if (mipMode == 0)
    {
      src = tex1D(tfTexture, sample);
    }
    else if ((mipMode == 1) && (sample > maxIntensity))
    {
      dst = tex1D(tfTexture, sample);
      maxIntensity = sample;
    }
    else if ((mipMode == 2) && (sample  < minIntensity))
    {
      dst = tex1D(tfTexture, sample);
      minIntensity = sample;
    }

    // Local illumination.
    if (lighting && (src.w > 0.1))
    {
      const float3 Ka = make_float3(0.0f, 0.0f, 0.0f);
      const float3 Kd = make_float3(0.8f, 0.8f, 0.8f);
      const float3 Ks = make_float3(0.8f, 0.8f, 0.8f);
      const float shininess = 1000.0f;
      if (justClippedPlane)
      {
        src = blinnPhong<bpc>(src, texCoord, L, H, Ka, Kd, Ks, shininess, &planeNormal);
        justClippedPlane = false;
      }
      else if (justClippedSphere)
      {
        float3 sphereNormal = normalize(pos - sphereCenter);
        src = blinnPhong<bpc>(src, texCoord, L, H, Ka, Kd, Ks, shininess, &sphereNormal);
        justClippedSphere = false;
      }
      else
      {
        src = blinnPhong<bpc>(src, texCoord, L, H, Ka, Kd, Ks, shininess);
      }
    }
    justClippedPlane = false;
    justClippedSphere = false;

    if (opacityCorrection)
    {
      src.w = 1 - powf(1 - src.w, dist);
    }

    // pre-multiply alpha
    src.x *= src.w;
    src.y *= src.w;
    src.z *= src.w;

    if (frontToBack && (mipMode == 0))
    {
      dst = dst + src * (1.0f - dst.w);
    }
    else if (!frontToBack && (mipMode == 0))
    {
      //dst = dst * src.w + src * (1.0f - src.w);
    }

    if (earlyRayTermination && (dst.w > opacityThreshold))
    {
      break;
    }

    t += tstep;
    if (t > tfar)
    {
      break;
    }

    pos += step;
  }
  d_output[y * width + x] = rgbaFloatToInt(dst);
}

vvRayRend::vvRayRend(vvVolDesc* vd, vvRenderState renderState)
  : vvRenderer(vd, renderState)
{
  glewInit();
  cudaGLSetGLDevice(0);

  _earlyRayTermination = true;
  _illumination = false;
  _interpolation = true;
  _opacityCorrection = true;

  _pbo = NULL;
  _gltex = NULL;

  initPbo(512, 512);

  d_randArray = 0;
  initRandTexture();

  d_volumeArray = 0;
  initVolumeTexture();

  d_transferFuncArray = 0;
  updateTransferFunction();
}

vvRayRend::~vvRayRend()
{
  cudaFreeArray(d_volumeArray);
  cudaFreeArray(d_transferFuncArray);
  cudaFreeArray(d_randArray);
}

int vvRayRend::getLUTSize() const
{
   vvDebugMsg::msg(2, "vvSoftVR::getLUTSize()");
   return (vd->getBPV()==2) ? 4096 : 256;
}

void vvRayRend::updateTransferFunction()
{
  int lutEntries = getLUTSize();
  float* rgba = new float[4 * lutEntries];

  vd->computeTFTexture(lutEntries, 1, 1, rgba);

  cudaChannelFormatDesc channelDesc = cudaCreateChannelDesc<float4>();

  cudaFreeArray(d_transferFuncArray);
  cudaMallocArray(&d_transferFuncArray, &channelDesc, lutEntries, 1);
  cudaMemcpyToArray(d_transferFuncArray, 0, 0, rgba, lutEntries * 4 * sizeof(float), cudaMemcpyHostToDevice);


  tfTexture.filterMode = cudaFilterModeLinear;
  tfTexture.normalized = true;    // access with normalized texture coordinates
  tfTexture.addressMode[0] = cudaAddressModeClamp;   // wrap texture coordinates

  cudaBindTextureToArray(tfTexture, d_transferFuncArray, channelDesc);

  delete[] rgba;
}

void vvRayRend::resize(const int width, const int height)
{
  initPbo(width, height);
  renderVolumeGL();
}

void vvRayRend::renderVolumeGL()
{
  vvDebugMsg::msg(1, "vvRayRend::renderVolumeGL()");

  const vvGLTools::Viewport vp = vvGLTools::getViewport();
  const int width = vp[2];
  const int height = vp[3];

  uint* d_output = 0;
  // map PBO to get CUDA device pointer
  cudaGLMapBufferObject((void**)&d_output, _pbo);

  dim3 blockSize(16, 16);
  dim3 gridSize = dim3(iDivUp(width, blockSize.x), iDivUp(height, blockSize.y));

  const vvVector3 size(vd->getSize());
  const vvVector3 size2 = size * 0.5f;

  // Assume: no probe.
  vvVector3 probeSizeObj;
  probeSizeObj.copy(&size);
  vvVector3 probeMin;
  probeMin = -size2;
  vvVector3 probeMax;
  probeMax = size2;

  vvVector3 clippedProbeSizeObj;
  clippedProbeSizeObj.copy(&probeSizeObj);
  for (int i=0; i<3; ++i)
  {
    if (clippedProbeSizeObj[i] < vd->getSize()[i])
    {
      clippedProbeSizeObj[i] = vd->getSize()[i];
    }
  }

  const float diagonal = sqrtf(clippedProbeSizeObj[0] * clippedProbeSizeObj[0] +
                               clippedProbeSizeObj[1] * clippedProbeSizeObj[1] +
                               clippedProbeSizeObj[2] * clippedProbeSizeObj[2]);

  const float diagonalVoxels = sqrtf(float(vd->vox[0] * vd->vox[0] +
                                           vd->vox[1] * vd->vox[1] +
                                           vd->vox[2] * vd->vox[2]));
  int numSlices = max(1, static_cast<int>(_renderState._quality * diagonalVoxels));

  // Inverse modelview-projection matrix.
  vvMatrix mvp, pr;
  getModelviewMatrix(&mvp);

  // Not related.
  vvMatrix invMV;
  invMV.copy(&mvp);
  invMV.invert();
  // Not related.

  getProjectionMatrix(&pr);
  mvp.multiplyPost(&pr);
  mvp.invert();

  float* viewM = new float[16];
  mvp.get(viewM);
  cudaMemcpyToSymbol(c_invViewMatrix, viewM, sizeof(float4) * 4);
  delete[] viewM;

  float3 volSize = make_float3(vd->vox[0], vd->vox[1], vd->vox[2]);

  bool isOrtho = pr.isProjOrtho();

  vvVector3 eye;
  getEyePosition(&eye);
  eye.multiply(&invMV);

  vvVector3 origin;

  vvVector3 normal;
  getObjNormal(normal, origin, eye, invMV, isOrtho);

  const float3 N = make_float3(normal[0], normal[1], normal[2]);

  const float3 L(-N);

  // Viewing direction.
  const float3 V(-N);

  // Half way vector.
  const float3 H = normalize(L + V);

  // Clip sphere.
  const float3 center = make_float3(_renderState._roiPos[0],
                                    _renderState._roiPos[1],
                                    _renderState._roiPos[2]);//make_float3(0.0f, 128.0f, 128.0f);
  const float radius = _renderState._roiSize[0] * vd->getSize()[0];//150;

  // Clip plane.
  const float3 pnormal = normalize(make_float3(0.0f, 0.71f, 0.63f));
  const float pdist = 0.0f;

  if (vd->bpc == 1)
  {
    if (_illumination && _earlyRayTermination && _opacityCorrection)
    {
      render<
             true, // Early ray termination.
             true, // Front to back.
             1, // Bytes per channel.
             0, // Mip mode.
             true, // Local illumination.
             true, // Opacity correction.
             false, // Jittering.
             false, // Clip sphere.
             false, // Clip plane.
             false // Show what's inside the clip sphere.
            ><<<gridSize, blockSize>>>(d_output, width, height,
                                       diagonalVoxels / (float)numSlices,
                                       volSize * 0.5f,
                                       L, H,
                                       center, radius * radius,
                                       pnormal, pdist);
    }
    else if (!_illumination && _earlyRayTermination && _opacityCorrection)
    {
      render<
             true, // Early ray termination.
             true, // Front to back.
             1, // Bytes per channel.
             0, // Mip mode.
             false, // Local illumination.
             true, // Opacity correction.
             false, // Jittering.
             false, // Clip sphere.
             false, // Clip plane.
             false // Show what's inside the clip sphere.
            ><<<gridSize, blockSize>>>(d_output, width, height,
                                       diagonalVoxels / (float)numSlices,
                                       volSize * 0.5f,
                                       L, H,
                                       center, radius * radius,
                                       pnormal, pdist);
    }
    else if (_illumination && !_earlyRayTermination && _opacityCorrection)
    {
      render<
             false, // Early ray termination.
             true, // Front to back.
             1, // Bytes per channel.
             0, // Mip mode.
             true, // Local illumination.
             true, // Opacity correction.
             false, // Jittering.
             false, // Clip sphere.
             false, // Clip plane.
             false // Show what's inside the clip sphere.
            ><<<gridSize, blockSize>>>(d_output, width, height,
                                       diagonalVoxels / (float)numSlices,
                                       volSize * 0.5f,
                                       L, H,
                                       center, radius * radius,
                                       pnormal, pdist);
    }
    else if (_illumination && _earlyRayTermination && !_opacityCorrection)
    {
      render<
             true, // Early ray termination.
             true, // Front to back.
             1, // Bytes per channel.
             0, // Mip mode.
             true, // Local illumination.
             false, // Opacity correction.
             false, // Jittering.
             false, // Clip sphere.
             false, // Clip plane.
             false // Show what's inside the clip sphere.
            ><<<gridSize, blockSize>>>(d_output, width, height,
                                       diagonalVoxels / (float)numSlices,
                                       volSize * 0.5f,
                                       L, H,
                                       center, radius * radius,
                                       pnormal, pdist);
    }
    else if (!_illumination && !_earlyRayTermination && _opacityCorrection)
    {
      render<
             false, // Early ray termination.
             true, // Front to back.
             1, // Bytes per channel.
             0, // Mip mode.
             false, // Local illumination.
             true, // Opacity correction.
             false, // Jittering.
             false, // Clip sphere.
             false, // Clip plane.
             false // Show what's inside the clip sphere.
            ><<<gridSize, blockSize>>>(d_output, width, height,
                                       diagonalVoxels / (float)numSlices,
                                       volSize * 0.5f,
                                       L, H,
                                       center, radius * radius,
                                       pnormal, pdist);
    }
    else if (_illumination && !_earlyRayTermination && !_opacityCorrection)
    {
      render<
             false, // Early ray termination.
             true, // Front to back.
             1, // Bytes per channel.
             0, // Mip mode.
             true, // Local illumination.
             false, // Opacity correction.
             false, // Jittering.
             false, // Clip sphere.
             false, // Clip plane.
             false // Show what's inside the clip sphere.
            ><<<gridSize, blockSize>>>(d_output, width, height,
                                       diagonalVoxels / (float)numSlices,
                                       volSize * 0.5f,
                                       L, H,
                                       center, radius * radius,
                                       pnormal, pdist);
    }
    else if (!_illumination && _earlyRayTermination && !_opacityCorrection)
    {
      render<
             true, // Early ray termination.
             true, // Front to back.
             1, // Bytes per channel.
             0, // Mip mode.
             false, // Local illumination.
             false, // Opacity correction.
             false, // Jittering.
             false, // Clip sphere.
             false, // Clip plane.
             false // Show what's inside the clip sphere.
            ><<<gridSize, blockSize>>>(d_output, width, height,
                                       diagonalVoxels / (float)numSlices,
                                       volSize * 0.5f,
                                       L, H,
                                       center, radius * radius,
                                       pnormal, pdist);
    }
    else if (!_illumination && !_earlyRayTermination && !_opacityCorrection)
    {
      render<
             false, // Early ray termination.
             true, // Front to back.
             1, // Bytes per channel.
             0, // Mip mode.
             false, // Local illumination.
             false, // Opacity correction.
             false, // Jittering.
             false, // Clip sphere.
             false, // Clip plane.
             false // Show what's inside the clip sphere.
            ><<<gridSize, blockSize>>>(d_output, width, height,
                                       diagonalVoxels / (float)numSlices,
                                       volSize * 0.5f,
                                       L, H,
                                       center, radius * radius,
                                       pnormal, pdist);
    }
  }
  else if (vd->bpc == 2)
  {
    if (_illumination && _earlyRayTermination && _opacityCorrection)
    {
      render<
             true, // Early ray termination.
             true, // Front to back.
             2, // Bytes per channel.
             0, // Mip mode.
             true, // Local illumination.
             true, // Opacity correction.
             false, // Jittering.
             false, // Clip sphere.
             false, // Clip plane.
             false // Show what's inside the clip sphere.
            ><<<gridSize, blockSize>>>(d_output, width, height,
                                       diagonalVoxels / (float)numSlices,
                                       volSize * 0.5f,
                                       L, H,
                                       center, radius * radius,
                                       pnormal, pdist);
    }
    else if (!_illumination && _earlyRayTermination && _opacityCorrection)
    {
      render<
             true, // Early ray termination.
             true, // Front to back.
             2, // Bytes per channel.
             0, // Mip mode.
             false, // Local illumination.
             true, // Opacity correction.
             false, // Jittering.
             false, // Clip sphere.
             false, // Clip plane.
             false // Show what's inside the clip sphere.
            ><<<gridSize, blockSize>>>(d_output, width, height,
                                       diagonalVoxels / (float)numSlices,
                                       volSize * 0.5f,
                                       L, H,
                                       center, radius * radius,
                                       pnormal, pdist);
    }
    else if (_illumination && !_earlyRayTermination && _opacityCorrection)
    {
      render<
             false, // Early ray termination.
             true, // Front to back.
             2, // Bytes per channel.
             0, // Mip mode.
             true, // Local illumination.
             true, // Opacity correction.
             false, // Jittering.
             false, // Clip sphere.
             false, // Clip plane.
             false // Show what's inside the clip sphere.
            ><<<gridSize, blockSize>>>(d_output, width, height,
                                       diagonalVoxels / (float)numSlices,
                                       volSize * 0.5f,
                                       L, H,
                                       center, radius * radius,
                                       pnormal, pdist);
    }
    else if (_illumination && _earlyRayTermination && !_opacityCorrection)
    {
      render<
             true, // Early ray termination.
             true, // Front to back.
             2, // Bytes per channel.
             0, // Mip mode.
             true, // Local illumination.
             false, // Opacity correction.
             false, // Jittering.
             false, // Clip sphere.
             false, // Clip plane.
             false // Show what's inside the clip sphere.
            ><<<gridSize, blockSize>>>(d_output, width, height,
                                       diagonalVoxels / (float)numSlices,
                                       volSize * 0.5f,
                                       L, H,
                                       center, radius * radius,
                                       pnormal, pdist);
    }
    else if (!_illumination && !_earlyRayTermination && _opacityCorrection)
    {
      render<
             false, // Early ray termination.
             true, // Front to back.
             2, // Bytes per channel.
             0, // Mip mode.
             false, // Local illumination.
             true, // Opacity correction.
             false, // Jittering.
             false, // Clip sphere.
             false, // Clip plane.
             false // Show what's inside the clip sphere.
            ><<<gridSize, blockSize>>>(d_output, width, height,
                                       diagonalVoxels / (float)numSlices,
                                       volSize * 0.5f,
                                       L, H,
                                       center, radius * radius,
                                       pnormal, pdist);
    }
    else if (_illumination && !_earlyRayTermination && !_opacityCorrection)
    {
      render<
             false, // Early ray termination.
             true, // Front to back.
             2, // Bytes per channel.
             0, // Mip mode.
             true, // Local illumination.
             false, // Opacity correction.
             false, // Jittering.
             false, // Clip sphere.
             false, // Clip plane.
             false // Show what's inside the clip sphere.
            ><<<gridSize, blockSize>>>(d_output, width, height,
                                       diagonalVoxels / (float)numSlices,
                                       volSize * 0.5f,
                                       L, H,
                                       center, radius * radius,
                                       pnormal, pdist);
    }
    else if (!_illumination && _earlyRayTermination && !_opacityCorrection)
    {
      render<
             true, // Early ray termination.
             true, // Front to back.
             2, // Bytes per channel.
             0, // Mip mode.
             false, // Local illumination.
             false, // Opacity correction.
             false, // Jittering.
             false, // Clip sphere.
             false, // Clip plane.
             false // Show what's inside the clip sphere.
            ><<<gridSize, blockSize>>>(d_output, width, height,
                                       diagonalVoxels / (float)numSlices,
                                       volSize * 0.5f,
                                       L, H,
                                       center, radius * radius,
                                       pnormal, pdist);
    }
    else if (!_illumination && !_earlyRayTermination && !_opacityCorrection)
    {
      render<
             false, // Early ray termination.
             true, // Front to back.
             2, // Bytes per channel.
             0, // Mip mode.
             false, // Local illumination.
             false, // Opacity correction.
             false, // Jittering.
             false, // Clip sphere.
             false, // Clip plane.
             false // Show what's inside the clip sphere.
            ><<<gridSize, blockSize>>>(d_output, width, height,
                                       diagonalVoxels / (float)numSlices,
                                       volSize * 0.5f,
                                       L, H,
                                       center, radius * radius,
                                       pnormal, pdist);
    }
  }
  cudaGLUnmapBufferObject(_pbo);

  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, _pbo);
  glBindTexture(GL_TEXTURE_2D, _gltex);
  glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

  renderQuad(width, height);
}

//----------------------------------------------------------------------------
// see parent
void vvRayRend::setParameter(const ParameterType param, const float newValue, char*)
{
  vvDebugMsg::msg(3, "vvTexRend::setParameter()");

  switch (param)
  {
  case vvRenderer::VV_SLICEINT:
    {
      const bool newInterpol = static_cast<bool>(newValue);
      if (_interpolation != newInterpol)
      {
        _interpolation = newInterpol;
        initVolumeTexture();
        updateTransferFunction();
      }
    }
    break;
  case vvRenderer::VV_LIGHTING:
    _illumination = static_cast<bool>(newValue);
    break;
  case vvRenderer::VV_OPCORR:
    _opacityCorrection = static_cast<bool>(newValue);
    break;
  default:
    vvRenderer::setParameter(param, newValue);
    break;
  }
}

void vvRayRend::initPbo(const int width, const int height)
{
  const int bitsPerPixel = 4;
  const int pboSize = width * height * bitsPerPixel;
  const int bufferSize = sizeof(GLubyte) * pboSize;
  GLubyte* pboSrc = new GLubyte[pboSize];
  glGenBuffers(1, &_pbo);
  glBindBuffer(GL_ARRAY_BUFFER, _pbo);
  glBufferData(GL_ARRAY_BUFFER, bufferSize, pboSrc, GL_DYNAMIC_DRAW);
  delete[] pboSrc;
  glBindBuffer(GL_ARRAY_BUFFER, 0);
  cudaGLRegisterBufferObject(_pbo);

  glGenTextures(1, &_gltex);
  glBindTexture(GL_TEXTURE_2D, _gltex);

  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

  glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA,
               GL_UNSIGNED_BYTE, NULL);
}

void vvRayRend::initRandTexture()
{
  const float scale = 2.0f;

  //srand(time(NULL));

  float4* randVecs = new float4[NUM_RAND_VECS];
  for (int i=0; i<NUM_RAND_VECS; ++i)
  {
    randVecs[i].x = (static_cast<float>(rand()) / static_cast<float>(INT_MAX)) * scale;
    randVecs[i].y = (static_cast<float>(rand()) / static_cast<float>(INT_MAX)) * scale;
    randVecs[i].z = (static_cast<float>(rand()) / static_cast<float>(INT_MAX)) * scale;
  }

  cudaChannelFormatDesc channelDesc = cudaCreateChannelDesc<float4>();

  cudaFreeArray(d_randArray);
  cudaMallocArray(&d_randArray, &channelDesc, NUM_RAND_VECS, 1);
  cudaMemcpyToArray(d_randArray, 0, 0, randVecs, NUM_RAND_VECS * sizeof(float4), cudaMemcpyHostToDevice);

  randTexture.filterMode = cudaFilterModeLinear;
  randTexture.addressMode[0] = cudaAddressModeClamp;

  cudaBindTextureToArray(randTexture, d_randArray, channelDesc);

  delete[] randVecs;
}

void vvRayRend::initVolumeTexture()
{
  cudaExtent volumeSize = make_cudaExtent(vd->vox[0], vd->vox[1], vd->vox[2]);

  cudaChannelFormatDesc channelDesc;
  if (vd->bpc == 1)
  {
    channelDesc = cudaCreateChannelDesc<uchar>();
  }
  else if (vd->bpc == 2)
  {
    channelDesc = cudaCreateChannelDesc<ushort>();
  }
  cudaMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize);

  cudaMemcpy3DParms copyParams = { 0 };

  if (vd->bpc == 1)
  {
    copyParams.srcPtr = make_cudaPitchedPtr(vd->getRaw(0), volumeSize.width*vd->bpc, volumeSize.width, volumeSize.height);
  }
  else if (vd->bpc == 2)
  {
    const int size = vd->vox[0] * vd->vox[1] * vd->vox[2] * vd->bpc;
    uchar* raw = vd->getRaw(0);
    uchar* data = new uchar[size];

    for (int i=0; i<size; i+=2)
    {
      int val = ((int) raw[i] << 8) | (int) raw[i + 1];
      val >>= 4;
      data[i] = raw[i];
      data[i + 1] = val;
    }
    copyParams.srcPtr = make_cudaPitchedPtr(data, volumeSize.width*vd->bpc, volumeSize.width, volumeSize.height);
  }
  copyParams.dstArray = d_volumeArray;
  copyParams.extent = volumeSize;
  copyParams.kind = cudaMemcpyHostToDevice;
  cudaMemcpy3D(&copyParams);

  if (vd->bpc == 1)
  {
      volTexture8.normalized = true;
      if (_interpolation)
      {
        volTexture8.filterMode = cudaFilterModeLinear;
      }
      else
      {
        volTexture8.filterMode = cudaFilterModePoint;
      }
      volTexture8.addressMode[0] = cudaAddressModeClamp;
      volTexture8.addressMode[1] = cudaAddressModeClamp;
      cudaBindTextureToArray(volTexture8, d_volumeArray, channelDesc);
  }
  else if (vd->bpc == 2)
  {
      volTexture16.normalized = true;
      if (_interpolation)
      {
        volTexture16.filterMode = cudaFilterModeLinear;
      }
      else
      {
        volTexture16.filterMode = cudaFilterModePoint;
      }
      volTexture16.addressMode[0] = cudaAddressModeClamp;
      volTexture16.addressMode[1] = cudaAddressModeClamp;
      cudaBindTextureToArray(volTexture16, d_volumeArray, channelDesc);
  }
}

void vvRayRend::renderQuad(const int width, const int height) const
{
  glDisable(GL_DEPTH_TEST);
  glDisable(GL_LIGHTING);
  glEnable(GL_TEXTURE_2D);
  glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE);

  glMatrixMode(GL_PROJECTION);
  glPushMatrix();
  glLoadIdentity();
  glOrtho(-1.0, 1.0, -1.0, 1.0, -1.0, 1.0);

  glMatrixMode( GL_MODELVIEW);
  glLoadIdentity();

  glViewport(0, 0, width, height);

  glClear(GL_COLOR_BUFFER_BIT);
  glBegin(GL_QUADS);
    glTexCoord2f(0.0, 0.0); glVertex3f(-1.0, -1.0, 0.0);
    glTexCoord2f(1.0, 0.0); glVertex3f(1.0, -1.0, 0.0);
    glTexCoord2f(1.0, 1.0); glVertex3f(1.0, 1.0, 0.0);
    glTexCoord2f(0.0, 1.0); glVertex3f(-1.0, 1.0, 0.0);
  glEnd();

  glMatrixMode(GL_PROJECTION);
  glPopMatrix();

  glDisable(GL_TEXTURE_2D);
}

#endif
